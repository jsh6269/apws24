
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_world() {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  printf("Device(GPU) Thread %d: Hello, World!\n", tidx);
}

int main()
{
  hello_world<<<3, 5>>>();
  hipDeviceSynchronize();
  return 0;
}
